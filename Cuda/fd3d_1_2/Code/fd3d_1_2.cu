#include "hip/hip_runtime.h"
//Calculus of the electromagnetic field using the FDTD method in Cuda with boundary condition so we have no reflected wave
//Juan Jos� Salazar

//Libraries
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

//Electromagnetic vectors dimension
#define dim 200

//Kernel function to calculate the fields
__global__ void fdtd(float* ex, float* hy, float* aux, int dimension, float puls, int mitad)
{
	//Wire id generation
	int id = threadIdx.x;





	if (id > 0)
	{
		//Ex vector calculus
		ex[id] = ex[id] + 0.5 * (hy[id - 1] - hy[id]);
	}




	ex[mitad] = puls;
	ex[0] = aux[0];
	aux[0] = ex[1];
	ex[dimension - 1] = aux[1];
	aux[1] = ex[dimension - 2];



	if (id < dimension - 1)
	{
		//Hy vector calculus
		hy[id] = hy[id] + 0.5 * (ex[id] - ex[id + 1]);

	}




}


//MAin function
int main(int argc, char** argv)
{
	//Declaration of host and device pointers
	float* hst_e, * hst_h, *hst_aux;
	float* dev_e, * dev_h, *dev_aux;
	//Block declaration
	int bloques = 1;
	//Pulse parameter declaration
	float pulse;
	int kc = (int)dim / 2;
	float t0 = 40.0;
	float spread = 12.0;
	float a;
	//Number of steps done
	int nsteps = 250;

	//Output files declaration (each one to store the fields values)
	ofstream ex;
	ofstream hy;

	//File opening
	ex.open("ex.txt");
	hy.open("hy.txt");

	//Host and device memory reserve
	hst_e = (float*)malloc(dim * sizeof(float));
	hst_h = (float*)malloc(dim * sizeof(float));
	hst_aux = (float*)malloc(2 * sizeof(float));

	hipMalloc(&dev_e, dim * sizeof(float));
	hipMalloc(&dev_h, dim * sizeof(float));
	hipMalloc(&dev_aux, 2 * sizeof(float));

	//Vectors initialization
	for (int i = 0;i < dim;i++)
	{
		hst_e[i] = 0;
		hst_h[i] = 0;
	}

	//Auxiliar vector for boundary conditions initialization
	hst_aux[0] = 0;
	hst_aux[1] = 0;

	//Host to device information movement
	hipMemcpy(dev_e, hst_e, dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_h, hst_h, dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_aux, hst_aux, 2 * sizeof(float), hipMemcpyHostToDevice);

	for (int i = 1;i < nsteps+1;i++)
	{
		//Pulse generation
		a = ((t0 - i) / spread) * ((t0 - i) / spread);
		a = -0.5 * a;
		pulse = expf(a);
		//cout << pulse << endl;
		//Kernel function call to use 200 wires (vectors dimension) to calculate the fields
		fdtd << <bloques, dim >> > (dev_e, dev_h, dev_aux, dim, pulse, kc);
	}



	//Device to host information movement
	hipMemcpy(hst_e, dev_e, dim * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hst_h, dev_h, dim * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hst_aux, dev_aux, 2 * sizeof(float), hipMemcpyDeviceToHost);




	//Results writing 
	for (int i = 0;i < dim;i++)
	{
		ex << hst_e[i] << endl;
		hy << hst_h[i] << endl;
		cout << hst_e[i] << endl;
	}


	// Device and host memory release
	hipFree(dev_e);
	hipFree(dev_h);
	hipFree(dev_aux);

	free(hst_e);
	free(hst_h);
	free(hst_aux);

	//File closing
	ex.close();
	hy.close();


	return 0;
}


